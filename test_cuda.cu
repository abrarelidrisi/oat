#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    
    printf("CUDA Device Count: %d\n", deviceCount);
    
    if (deviceCount > 0) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        printf("Device 0: %s\n", prop.name);
        printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
        printf("Total Global Memory: %.2f GB\n", prop.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
    }
    
    return 0;
}
